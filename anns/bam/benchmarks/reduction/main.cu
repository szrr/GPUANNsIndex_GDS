#include "hip/hip_runtime.h"
/* References:
 *
 *      Coalesce
 *          Hong, Sungpack, et al.
 *          "Accelerating CUDA graph algorithms at maximum warp."
 *          Acm Sigplan Notices 46.8 (2011): 267-276.
 *
 */

#include <hip/hip_runtime.h>
#include <fstream>
#include <stdint.h>
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <getopt.h>
//#include "hip/hip_runtime_api.h"
#include <algorithm>
#include <vector>
#include <numeric>
#include <iterator>
#include <math.h>
#include <chrono>
#include <ctime>
#include <ratio>
#include <cstdio>
#include <cstdint>
#include <cstring>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <stdexcept>

#include <nvm_ctrl.h>
#include <nvm_types.h>
#include <nvm_queue.h>
#include <nvm_util.h>
#include <nvm_admin.h>
#include <nvm_error.h>
#include <nvm_cmd.h>
#include <buffer.h>
#include "settings.h"
#include <ctrl.h>
#include <event.h>
#include <queue.h>
#include <nvm_parallel_queue.h>
#include <nvm_io.h>
#include <page_cache.h>
#include <util.h>

#include <iterator> 
#include <numeric> 
#include <functional>
#include <hipcub/hipcub.hpp>


#define UINT64MAX 0xFFFFFFFFFFFFFFFF

using error = std::runtime_error;
using std::string;
//const char* const ctrls_paths[] = {"/dev/libnvmpro0", "/dev/libnvmpro1", "/dev/libnvmpro2", "/dev/libnvmpro3", "/dev/libnvmpro4", "/dev/libnvmpro5", "/dev/libnvmpro6", "/dev/libnvmpro7"};
//const char* const ctrls_paths[] = {"/dev/libnvm0", "/dev/libnvm1", "/dev/libnvm2", "/dev/libnvm3", "/dev/libnvm4", "/dev/libnvm5", "/dev/libnvm6", "/dev/libnvm7", "/dev/libnvm8", "/dev/libnvm9"};
const char* const ctrls_paths[] = {"/dev/libnvm0", "/dev/libnvm1", "/dev/libnvm4", "/dev/libnvm9", "/dev/libnvm2", "/dev/libnvm3", "/dev/libnvm5", "/dev/libnvm6", "/dev/libnvm7", "/dev/libnvm8"};

#define WARP_SHIFT 5
#define WARP_SIZE 32

#define CHUNK_SHIFT 3
#define CHUNK_SIZE (1 << CHUNK_SHIFT)

#define BLOCK_NUM 1024ULL

#define MAXWARP 64

typedef uint64_t ElemT;

typedef enum {
    BASELINE = 0,
    OPTIMIZED=1,
    BASELINE_PC = 2,
    OPTIMIZED_PC=3,
} impl_type;

typedef enum {
    GPUMEM = 0,
    UVM_READONLY = 1,
    UVM_DIRECT = 2,
    BAFS_DIRECT= 6,
} mem_type;



__global__ 
void kernel_reduce_baseline(ElemT *input, ElemT *output, uint64_t len){

    extern __shared__ ElemT sharedMem[]; 

    uint64_t tid = threadIdx.x; 
    uint64_t start = 2*blockDim.x*blockIdx.x; 

    if((start+tid) < len)
        sharedMem[tid] = input[start+tid];
    else 
        sharedMem[tid] = 0; 

    if((blockDim.x+start+tid) < len)
        sharedMem[blockDim.x+tid] = input[blockDim.x+start+tid];
    else 
        sharedMem[blockDim.x+tid] = 0; 

    for(uint64_t stride = blockDim.x; stride>0; stride >>=1){
        __syncthreads();

        if( tid < stride)
            sharedMem[tid] += sharedMem[stride+tid];
    }

    if(threadIdx.x == 0)
        output[blockIdx.x] = sharedMem[0];

}

__global__ 
void kernel_reduce_optimized(ElemT *input, ElemT *output, uint64_t len, uint64_t page_size, uint64_t n_warps){
    extern __shared__ ElemT sharedMem[];

    uint64_t tid = blockDim.x*blockIdx.x+threadIdx.x; 
    uint64_t laneid = tid % WARP_SIZE;
    uint64_t warp_id = tid / WARP_SIZE;
    uint64_t num_elems_per_cl = page_size/sizeof(ElemT); 
    
    uint64_t start = 2*warp_id * num_elems_per_cl;  

    for(size_t i=laneid; i<num_elems_per_cl;i+=WARP_SIZE){

        if(((start+i)<len))
            sharedMem[i] = input[start+i];
        else 
            sharedMem[i] =0;

        if((num_elems_per_cl+start+i)<len)
            sharedMem[num_elems_per_cl+i] = input[num_elems_per_cl+start+i];
        else 
            sharedMem[i] =0;

        __syncwarp(); 
    }

    for(uint64_t stride = num_elems_per_cl; stride>0; stride>>=1){
        __syncwarp(); 

        for(size_t i =0; i<num_elems_per_cl; i+=WARP_SIZE){
            uint64_t idx = i*WARP_SIZE + laneid; 
        
            if(idx <stride)
                sharedMem[idx] += sharedMem[stride+idx]; 
        }
    }

    if(laneid == 0)
        output[warp_id] = sharedMem[0];
}



__global__ 
void kernel_reduce_baseline_ptr_pc(array_d_t<ElemT> *da, ElemT *output, uint64_t len){

    extern __shared__ ElemT sharedMem[]; 

    bam_ptr<ElemT> input(da);

    uint64_t tid = threadIdx.x; 
    uint64_t start = 2*blockDim.x*blockIdx.x; 

    if((start+tid) < len)
        sharedMem[tid] = input[start+tid];
    else 
        sharedMem[tid] = 0; 

    if((blockDim.x+start+tid) < len)
        sharedMem[blockDim.x+tid] = input[blockDim.x+start+tid];
    else 
        sharedMem[blockDim.x+tid] = 0; 

    for(uint64_t stride = blockDim.x; stride>0; stride >>=1){
        __syncthreads();

        if( tid < stride)
            sharedMem[tid] += sharedMem[stride+tid];
    }

    if(threadIdx.x == 0)
        output[blockIdx.x] = sharedMem[0];

}

__global__ 
void kernel_reduce_optimized_ptr_pc(array_d_t<ElemT> *da, ElemT *output, uint64_t len, uint64_t page_size, uint64_t n_warps){
    extern __shared__ ElemT sharedMem[];
    bam_ptr<ElemT> input(da);

    uint64_t tid = blockDim.x*blockIdx.x+threadIdx.x; 
    uint64_t laneid = tid % WARP_SIZE;
    uint64_t warp_id = tid / WARP_SIZE;
    uint64_t num_elems_per_cl = page_size/sizeof(ElemT); 
    
    uint64_t start = 2*warp_id * num_elems_per_cl;  

    for(size_t i=laneid; i<num_elems_per_cl;i+=WARP_SIZE){

        if(((start+i)<len))
            sharedMem[i] = input[start+i];
        else 
            sharedMem[i] =0;

        if((num_elems_per_cl+start+i)<len)
            sharedMem[num_elems_per_cl+i] = input[num_elems_per_cl+start+i];
        else 
            sharedMem[i] =0;

        __syncwarp(); 
    }

    for(uint64_t stride = num_elems_per_cl; stride>0; stride>>=1){
        __syncwarp(); 

        for(size_t i =0; i<num_elems_per_cl; i+=WARP_SIZE){
            uint64_t idx = i*WARP_SIZE + laneid; 
        
            if(idx <stride)
                sharedMem[idx] += sharedMem[stride+idx]; 
        }
    }

    if(laneid == 0)
        output[warp_id] = sharedMem[0];
}

int main(int argc, char *argv[]) {
    using namespace std::chrono; 

    Settings settings; 
    try
    {
        settings.parseArguments(argc, argv);
    }
    catch (const string& e)
    {
        fprintf(stderr, "%s\n", e.c_str());
        fprintf(stderr, "%s\n", Settings::usageString(argv[0]).c_str());
        return 1;
    }

    hipDeviceProp_t properties;
    if (hipGetDeviceProperties(&properties, settings.cudaDevice) != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device properties\n");
        return 1;
    }

    std::ifstream filea, fileb;
    std::string a_file;
    std::string a_file_bin;
    std::string filename;

    impl_type type;
    mem_type mem;
    uint64_t *a_h, *a_d;
    uint64_t numblocks, numthreads;

    float milliseconds;

    uint64_t pc_page_size;
    uint64_t pc_pages; 

    try{

        a_file = std::string(settings.input_a); 
        
        type = (impl_type) settings.type; 
        mem = (mem_type) settings.memalloc; 

        pc_page_size = settings.pageSize; 
        pc_pages = ceil((float)settings.maxPageCacheSize/pc_page_size);

        numthreads = settings.numThreads;
        
        cuda_err_chk(hipSetDevice(settings.cudaDevice));
        
        hipEvent_t start, end, tstart, tend;
        cuda_err_chk(hipEventCreate(&start));
        cuda_err_chk(hipEventCreate(&end));
        cuda_err_chk(hipEventCreate(&tstart));
        cuda_err_chk(hipEventCreate(&tend));


        a_file_bin = a_file + ".dst";

        std::cout << "A: " << a_file_bin  << std::endl;

        uint64_t n_elems = settings.n_elems;
        uint64_t n_elems_size = n_elems * sizeof(uint64_t);
        printf("Total elements: %llu \n", n_elems);
        uint64_t tmp; 
        
        // Read files
        filea.open(a_file_bin.c_str(), std::ios::in | std::ios::binary);
        if (!filea.is_open()) {
            printf("A file open failed\n");
            exit(1);
        };

        filea.read((char*)(&tmp), 16);
        if(mem != BAFS_DIRECT)
            a_h = (uint64_t*)calloc(n_elems_size, sizeof(uint64_t));
        if((mem!=BAFS_DIRECT) &&  (mem != UVM_DIRECT)){
             filea.read((char*)a_h, n_elems_size);
             filea.close();
        }

        switch (mem) {
            case GPUMEM:
                {  
                cuda_err_chk(hipMalloc((void**)&a_d, n_elems_size));
                high_resolution_clock::time_point mc1 = high_resolution_clock::now();
                cuda_err_chk(hipMemcpy(a_d, a_h, n_elems_size, hipMemcpyHostToDevice));
                high_resolution_clock::time_point mc2 = high_resolution_clock::now();
                duration<double> mc_time_span = duration_cast<duration<double>>(mc2 -mc1);
                std::cout<< "Memcpy time for loading the inputs: "<< mc_time_span.count() <<std::endl;
                break;
                }
            case UVM_READONLY:
                {
                cuda_err_chk(hipMallocManaged((void**)&a_d, n_elems_size));
                cuda_err_chk(hipMemcpy(a_d, a_h, n_elems_size, hipMemcpyHostToDevice));
                cuda_err_chk(hipMemAdvise(a_d, n_elems_size, hipMemAdviseSetReadMostly, settings.cudaDevice));
                break;
                }
            case UVM_DIRECT:
                {
                filea.close();
                fileb.close();
                int fda = open(a_file_bin.c_str(), O_RDONLY | O_DIRECT); 
                FILE *fa_tmp= fdopen(fda, "rb");
                if ((fa_tmp == NULL) || (fda == -1)) {
                    printf("A file fd open failed\n");
                    exit(1);
                }   
                
                uint64_t count_4k_aligned = ((n_elems + 2 + 4096 / sizeof(uint64_t)) / (4096 / sizeof(uint64_t))) * (4096 / sizeof(uint64_t));
                //uint64_t count_4k_aligned = n_elems; 
                uint64_t size_4k_aligned = count_4k_aligned * sizeof(uint64_t);

                cuda_err_chk(hipMallocManaged((void**)&a_d, size_4k_aligned));
                cuda_err_chk(hipMemAdvise(a_d, size_4k_aligned, hipMemAdviseSetAccessedBy, settings.cudaDevice));
                high_resolution_clock::time_point ft1 = high_resolution_clock::now();
               
                if (fread(a_d, sizeof(uint64_t), count_4k_aligned, fa_tmp)) {
                    printf("A file fread failed: %llu \t %llu\n", count_4k_aligned, n_elems+2);
                    exit(1);
                }   
                fclose(fa_tmp);                                                                                                              
                close(fda);
                
                a_d = a_d + 2;

                high_resolution_clock::time_point ft2 = high_resolution_clock::now();
                duration<double> time_span = duration_cast<duration<double>>(ft2 -ft1);
                std::cout<< "file read time: "<< time_span.count() <<std::endl;
                
                break;
                }
            case BAFS_DIRECT: 
                {
                break;
                }
        }

        
        uint64_t n_pages = ceil(((float)n_elems_size)/pc_page_size); 

        uint64_t n_warps = 0; 

        switch (type) {
            case BASELINE:
            case BASELINE_PC:{
                numblocks = ((n_elems/(2*numthreads)) + 1);
                break;
            }
            case OPTIMIZED: 
            case OPTIMIZED_PC:{
                uint64_t num_elems_per_cl = pc_page_size/sizeof(ElemT); 
                n_warps = ceil(n_pages/2);
                numblocks = ceil(n_warps*WARP_SIZE/numthreads); 
                break;
            }

            default:
                fprintf(stderr, "Invalid type\n");
                exit(1);
                break;
        }
        
        //dim3 blockDim(BLOCK_NUM, (numblocks+BLOCK_NUM)/BLOCK_NUM);
        dim3 blockDim((numblocks));
        if((type == BASELINE_PC)) {
                printf("page size: %d, pc_entries: %llu\n", pc_page_size, pc_pages);
        }
        
        // Allocate memory for GPU
        ElemT *result_h;
        ElemT *result_d;
        ElemT *dev2out_d; 
        ElemT *int_d; 

        cuda_err_chk(hipMalloc((void**)&int_d, (numblocks)*sizeof(ElemT)));
        cuda_err_chk(hipMalloc((void**)&dev2out_d, (numblocks)*sizeof(ElemT)));
        cuda_err_chk(hipMalloc((void**)&result_d, (n_elems+1)*sizeof(ElemT)));
        result_h = (ElemT*) malloc(n_elems* sizeof(ElemT)); 
        cuda_err_chk(hipMemset(result_d, 0, (n_elems+1)*sizeof(ElemT)));

		printf("Allocation finished\n");
        fflush(stdout);

        std::vector<Controller*> ctrls(settings.n_ctrls);
        if(mem == BAFS_DIRECT){
            cuda_err_chk(hipSetDevice(settings.cudaDevice));
            for (size_t i = 0 ; i < settings.n_ctrls; i++)
                ctrls[i] = new Controller(ctrls_paths[i], settings.nvmNamespace, settings.cudaDevice, settings.queueDepth, settings.numQueues);
            printf("Controllers Created\n");
        }
        printf("Initialization done\n");
        fflush(stdout);

        page_cache_t* h_pc;
        range_t<uint64_t>* h_Arange;
        std::vector<range_t<uint64_t>*> vec_Arange(1);
        array_t<uint64_t>* h_Aarray;


        if((type == BASELINE_PC) || (type == OPTIMIZED_PC)) {
            //TODO: fix for 2 arrays
            h_pc =new page_cache_t(pc_page_size, pc_pages, settings.cudaDevice, ctrls[0][0], (uint64_t) 64, ctrls);
            h_Arange = new range_t<uint64_t>((uint64_t)0 ,(uint64_t)n_elems, (uint64_t) (ceil(settings.afileoffset*1.0/pc_page_size)),(uint64_t)n_pages, (uint64_t)0, (uint64_t)pc_page_size, h_pc, settings.cudaDevice); 
            vec_Arange[0] = h_Arange; 
            h_Aarray = new array_t<uint64_t>(n_elems, settings.afileoffset, vec_Arange, settings.cudaDevice);

            printf("Page cache initialized\n");
            fflush(stdout);
        }

        void *d_tmp = NULL;
        size_t tmp_size =0; 

        for(int titr=0; titr<1; titr+=1){
            cuda_err_chk(hipEventRecord(start, 0));
                
            auto itrstart = std::chrono::system_clock::now();

            switch (type) {
                case BASELINE:{
                    printf("launching PC: blockDim.x :%llu blockDim.y :%llu numthreads:%llu sharedMemSize: %llu\n", blockDim.x, blockDim.y, numthreads, 2*numblocks*sizeof(ElemT));
                    kernel_reduce_baseline<<<blockDim, numthreads, 2*numthreads*sizeof(ElemT)>>>(a_d, result_d, n_elems);
                    break;
                }
                case OPTIMIZED:{
                    printf("launching PC: blockDim.x :%llu blockDim.y :%llu numthreads:%llu sharedMemSize: %llu\n", blockDim.x, blockDim.y, numthreads, 2*pc_page_size);
                    kernel_reduce_optimized<<<blockDim, numthreads, 2*pc_page_size>>>(a_d, result_d, n_elems, settings.pageSize, n_warps);
                    break;

                    }
                case BASELINE_PC:{

                    printf("launching PC: blockDim.x :%llu blockDim.y :%llu numthreads:%llu\n", blockDim.x, blockDim.y, numthreads);
                    kernel_reduce_baseline_ptr_pc<<<blockDim, numthreads, 2*numthreads*sizeof(ElemT)>>>(h_Aarray->d_array_ptr, result_d, n_elems);
                    break;
                }
                case OPTIMIZED_PC:{
                    printf("launching PC: blockDim.x :%llu blockDim.y :%llu numthreads:%llu sharedMemSize: %llu\n", blockDim.x, blockDim.y, numthreads, 2*pc_page_size);
                    kernel_reduce_optimized_ptr_pc<<<blockDim, numthreads, 2*pc_page_size>>>(h_Aarray->d_array_ptr, result_d, n_elems, settings.pageSize, n_warps);
                    break;
                }
                default:
                    fprintf(stderr, "Invalid type\n");
                    exit(1);
                    break;
            }
            cuda_err_chk(hipEventRecord(end, 0));
            cuda_err_chk(hipEventSynchronize(end));
            cuda_err_chk(hipEventElapsedTime(&milliseconds, start, end));
            
            cuda_err_chk(hipMemcpy(result_h, (result_d), (n_elems+1)*sizeof(ElemT), hipMemcpyDeviceToHost));
            //printf("\n******\n");
            //fflush(stdout);
            //if(mem != BAFS_DIRECT){
            //   printf("Input list::");
            //   for (uint64_t i=n_elems-100; i< n_elems; i++)
            //       printf("%llu\t", a_h[i]);
            //}
            //printf("\n\nScan result:");
            //for (uint64_t i=n_elems-100; i< n_elems; i++)
            //    printf("%llu\t", result_h[i]);
            //printf("\n******\n");
           
            //std::vector<uint64_t> a_h_vec (a_h, a_h+n_elems);
            //uint64_t total = std::accumulate(a_h_vec.begin(), a_h_vec.begin()+n_elems, 0, std::plus<uint64_t>());
            uint64_t total = 0;
            if(mem != BAFS_DIRECT){
                for(uint64_t count=0; count<n_elems; count++)
                    total+=a_h[count];
                printf("total in cpu: %llu \n", total);
            }

            total =0; 
            for(uint64_t count=0; count<numblocks ;count++){
                total += result_h[count];
            }
            printf("total in gpu: %llu \n ", total);
            auto itrend = std::chrono::system_clock::now();
            auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(itrend - itrstart);

            //if(mem == BAFS_DIRECT) {
            //         h_Aarray->print_reset_stats();
		    // printf("VA SSD: %d PageSize: %d itrTime: %f\n", settings.n_ctrls, settings.pageSize, (double)elapsed.count()); 
            //}



            if(mem == BAFS_DIRECT) {
                 h_Aarray->print_reset_stats();
                 cuda_err_chk(hipDeviceSynchronize());
            }
            printf("\nVA %d A:%s Impl: %d \t SSD: %d \t CL: %d \t Cache: %llu \t TotalTime %f ms\n", titr, a_file_bin.c_str(), type, settings.n_ctrls, settings.pageSize,settings.maxPageCacheSize, milliseconds); 
            fflush(stdout);
        }

        if(mem!=BAFS_DIRECT){
           free(a_h);
         }
        free(result_h);

        if((type == BASELINE_PC) || (type == OPTIMIZED_PC)) {
            //TODO: Fix this
            delete h_pc;
            delete h_Arange;
            delete h_Aarray;
        }

        if(mem!=BAFS_DIRECT){
            if(mem==UVM_DIRECT){
              a_d = a_d-2; 
            }
            cuda_err_chk(hipFree(a_d));
            cuda_err_chk(hipFree(int_d));
            cuda_err_chk(hipFree(dev2out_d));
            cuda_err_chk(hipFree(result_d));
        }
            
        for (size_t i = 0 ; i < settings.n_ctrls; i++)
             delete ctrls[i];

    }
    catch (const error& e){
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }

    return 0;
}
