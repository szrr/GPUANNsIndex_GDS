#include <hip/hip_runtime_api.h>
#include "DeviceUtils.h"
#include <mutex>
#include <unordered_map>

int getDeviceForAddress(const void* p) {
    if (!p) {
        return -1;
    }

    hipPointerAttribute_t att;
    hipError_t err = hipPointerGetAttributes(&att, p);

    if (err == hipErrorInvalidValue) {
        // Make sure the current thread error status has been reset
        err = hipGetLastError();
        return -1;
    }

    // FIXME: what to use for managed memory?
    if (att.type == hipMemoryTypeDevice) {
        return att.device;
    } else {
        return -1;
    }
}

// Helper function to get device properties and cache them
const hipDeviceProp_t& getDeviceProperties(int device) {
    static std::mutex mutex;
    static std::unordered_map<int, hipDeviceProp_t> properties;

    std::lock_guard<std::mutex> guard(mutex);

    auto it = properties.find(device);
    if (it == properties.end()) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device);

        properties[device] = prop;
        it = properties.find(device);
    }

    return it->second;
}

// Function to get the maximum number of threads per block for a given device
int getMaxThreads(int device) {
    return getDeviceProperties(device).maxThreadsPerBlock;
}